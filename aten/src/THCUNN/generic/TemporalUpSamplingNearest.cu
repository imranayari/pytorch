
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/TemporalUpSamplingNearest.cu"
#else

#include "../common.h"

static inline void THNN_(TemporalUpSamplingNearest_shapeCheck)
                        (THCState *state,THCTensor *input, THCTensor *gradOutput,
                         int scale_factor) {
  THArgCheck(input != NULL, 2, "3D input tensor expected but got NULL");
  THArgCheck(scale_factor > 1, 4,
             "scale_factor must be greater than 1, but got: %d", scale_factor);
  THCUNN_argCheck(state, input->nDimension == 2 || input->nDimension == 3, 2, input,
                  "2D or 3D input tensor expected but got: %s");
  if (input->nDimension == 2) {
    int nChannels    = THCTensor_(size)(state, input, 0);
    int inputWidth   = THCTensor_(size)(state, input, 1);
    int outputWidth  = inputWidth  * scale_factor;
    if (gradOutput != NULL) {
      THCUNN_check_dim_size(state, gradOutput, 2, 0, nChannels);
      THCUNN_check_dim_size(state, gradOutput, 2, 1, outputWidth);
    }
  } else {
    int nBatch       = THCTensor_(size)(state, input, 0);
    int nChannels    = THCTensor_(size)(state, input, 1);
    int inputWidth   = THCTensor_(size)(state, input, 2);
    int outputWidth  = inputWidth  * scale_factor;
    if (gradOutput != NULL) {
      THCUNN_check_dim_size(state, gradOutput, 3, 0, nBatch);
      THCUNN_check_dim_size(state, gradOutput, 3, 1, nChannels);
      THCUNN_check_dim_size(state, gradOutput, 3, 2, outputWidth);
    }
  }
}

void THNN_(TemporalUpSamplingNearest_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int scale_factor)
{
  THCTensor_(zero)(state, output);

  THCUNN_assertSameGPU(state, 2, input, output);
  THNN_(TemporalUpSamplingNearest_shapeCheck)(state, input, NULL, scale_factor);
  int inputWidth  = THCTensor_(size)(state, input,  input->nDimension-1);
  int outputWidth = inputWidth * scale_factor;

   if (input->nDimension == 2) {
     THCTensor_(resize2d)(state, output,
                          THCTensor_(size)(state, input, 0),
                          outputWidth);
   } else {
     THCTensor_(resize3d)(state, output,
                          THCTensor_(size)(state, input, 0),
                          THCTensor_(size)(state, input, 1),
                          outputWidth);
  }

  input = THCTensor_(newContiguous)(state, input);
  // This is for allocating output Tensor
  int64_t no_elements = 1;
  for(int i = 0; i < input->nDimension; i++){
    no_elements *= input->size[i];
  }
  no_elements *= scale_factor;

  int d1;
  int d2;

  if (input->nDimension == 2) {
    d1 = output->size[0];
    d2 = output->size[1];
  } else {
    d1 = output->size[1];
    d2 = output->size[2];
  }

  real *input_data = THCTensor_(data)(state, input);
  real *output_data = THCTensor_(data)(state, output);

  // cuda blocks & threads:
  int64_t nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  int64_t n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  int64_t n_yblocks = (int64_t)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  upscale<<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data, no_elements, scale_factor, d1, d2);
  THCudaCheck(cudaGetLastError());

  // final cut:
  THCTensor_(free)(state, input);
}

void THNN_(TemporalUpSamplingNearest_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int scale_factor)
{

  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  THNN_(TemporalUpSamplingNearest_shapeCheck)(state, input, gradOutput, scale_factor);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  THCTensor_(zero)(state, gradInput);

  real *gradInput_data = THCTensor_(data)(state, gradInput);
  real *gradOutput_data = THCTensor_(data)(state, gradOutput);

  int64_t no_elements = 1;
  for(int i = 0; i < gradInput->nDimension; i++){
    no_elements *= gradInput->size[i];
  }

  int d1;
  int d2;

  if (gradInput->nDimension == 2) {
    d1 = gradInput->size[0];
    d2 = gradInput->size[1];
  } else {
    d1 = gradInput->size[1];
    d2 = gradInput->size[2];
  }

  // cuda blocks & threads:
  int64_t nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  int64_t n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  int64_t n_yblocks = (int64_t)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  downscale<real ,accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data, no_elements,
    scale_factor, d1, d2);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, gradOutput);
}

#endif
